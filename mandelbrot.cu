
#include <hip/hip_runtime.h>
#include <cstdint>

// Algorithm parameters.
const float escape_radius = 2.5;
const int max_iterations = 60; // Must be divisible by 3.
const int blocks_size_x = 8;
const int blocks_size_y = 10;
const int threads_size_x = 32;
const int threads_size_y = 32;

typedef struct {
    uint8_t red;
    uint8_t green;
    uint8_t blue;
} color_t;

// Counts the number of iterations for the series to diverge.
__device__ uint8_t count_iterations(float x, float y) {
    uint8_t iterations = 0;
    float u = 0.0, v = 0.0;
    while (iterations < max_iterations && (u * u + v * v < escape_radius * escape_radius)) {
        float _u = u * u - v * v + x;
        float _v = 2 * u * v + y;
        u = _u;
        v = _v;
        iterations++;
    }
    return iterations;
}

// Linerarly scales the interval [0:size] to [start:stop].
__device__ float scale(int index, int size, float start, float stop)
{
    return (stop - start) * index / size + start;
}

// Paints the pixel at coordinates (x_index, y_index) of an image buffer.
__device__ void paint(uint8_t* buffer, int x_index, int y_index, int x_size, color_t color) {
    int index = 3 * (x_index + (y_index * x_size));
    buffer[index] = color.red;
    buffer[index+1] = color.green;
    buffer[index+2] = color.blue;
}

// Converts a number of iterations to a RGB color.
__device__ color_t palette(uint8_t iterations) {
    uint8_t red = 0, green = 0, blue = 0;
    if (iterations < max_iterations) {
        if (iterations < max_iterations / 3) {
            red = 255 * 3 * iterations / max_iterations;
        } else if (iterations < 2 * max_iterations / 3) {
            red = 255;
            green = 255 * 3 * (iterations - (max_iterations / 3)) / max_iterations;
        } else {
            red = 255;
            green = 255;
            blue = 255 * 3 * (iterations - (2 * max_iterations / 3)) / max_iterations;
        }
    }
    return { red, green, blue };
}

// Partially renders the image buffer for a single thread.
__global__ void render_thread(uint8_t *buffer,
    int x_size, float x_start, float x_stop,
    int y_size, float y_start, float y_stop)
{
    int y_first = threadIdx.y + (blockIdx.y * blockDim.y);
    int y_step = blockDim.y * gridDim.y;
    for (int y_index=y_first; y_index<y_size; y_index+=y_step) {
        float y = scale(y_index, y_size, y_start, y_stop);
        int x_first = threadIdx.x + (blockIdx.x * blockDim.x);
        int x_step = blockDim.x * gridDim.x;
        for (int x_index=x_first; x_index<x_size; x_index+=x_step) {
            float x = scale(x_index, x_size, x_start, x_stop);
            uint8_t iterations = count_iterations(x, y);
            color_t color = palette(iterations);
            paint(buffer, x_index, y_index, x_size, color);
        }
    }
}

// Renders the entire image buffer.
extern "C" void render(uint8_t *buffer,
    int x_size, float x_start, float x_stop,
    int y_size, float y_start, float y_stop)
{
    uint8_t* gpu_buffer;
    size_t size = sizeof(uint8_t) * 3 * x_size * y_size;
    hipMalloc(&gpu_buffer, size);
    render_thread<<<
        dim3(blocks_size_x, blocks_size_y),
        dim3(threads_size_x, threads_size_y)>>>(gpu_buffer,
            x_size, x_start, x_stop,
            y_size, y_start, y_stop);
    hipMemcpy(buffer, gpu_buffer, size, hipMemcpyDeviceToHost);
    hipFree(gpu_buffer);
}
